#include "hip/hip_runtime.h"


inline __device__ real velocityFluxX(real avg_neg,real avg_pos,real* cellParams,real* blockParams) {

   creal VY = blockParams[BlockParams::VYCRD] + (threadIdx.z+0.5f)*blockParams[BlockParams::DVY];
   creal VZ = blockParams[BlockParams::VZCRD] + (threadIdx.y+0.5f)*blockParams[BlockParams::DVZ];
   creal EX = cellParams[CellParams::EX];
   creal BY = cellParams[CellParams::BY];
   creal BZ = cellParams[CellParams::BZ];
   
   creal AX = blockParams[BlockParams::Q_PER_M]*(EX + VY*BZ - VZ*BY);
   return 0.5f*AX*(avg_neg + avg_pos) - 0.5f*fabsf(AX)*(avg_pos-avg_neg);
}

inline __device__ real velocityFluxY(real avg_neg,real avg_pos,real* cellParams,real* blockParams) {
   /*
    creal VX = blockParams[BlockParams::VXCRD] + (threadIdx.x+0.5f)*blockParams[BlockParams::DVX];
    creal VZ = blockParams[BlockParams::VZCRD] + (threadIdx.y+0.5f)*blockParams[BlockParams::DVZ];
    creal EY = cellParams[CellParams::EY];
    creal BX = cellParams[CellParams::BX];
    creal BZ = cellParams[CellParams::BZ];
    * 
    creal AY = blockParams[BlockParams::Q_PER_M]*(EY + VZ*BX - VX*BZ);
    return 0.5f*AY*(avg_neg + avg_pos) - 0.5f*fabsf(AY)*(avg_pos-avg_neg);
    */
   return 0.0f;
}

inline __device__ real velocityFluxZ(real avg_neg,real avg_pos,real* cellParams,real* blockParams) {
   /*
    creal VX = blockParams[BlockParams::VXCRD] + (threadIdx.x+0.5f)*blockParams[BlockParams::DVX];
    creal VY = blockParams[BlockParams::VYCRD] + (threadIdx.y+0.5f)*blockParams[BlockParams::DVY];
    creal EZ = cellParams[CellParams::EZ];
    creal BX = cellParams[CellParams::BX];
    creal BY = cellParams[CellParams::BY];
 
    creal AZ = blockParams[BlockParams::Q_PER_M]*(EZ + VX*BY - VY*BX);
    return 0.5f*AZ*(avg_neg + avg_pos) - 0.5f*fabsf(AZ)*(avg_pos-avg_neg);
    */
   return 0.0f;
}

inline __device__ real spatialFluxX(real avg_neg,real avg_pos,real* blockParams) {
   creal VX = blockParams[BlockParams::VXCRD] + (threadIdx.x+0.5f)*blockParams[BlockParams::DVX];
   return 0.5f*VX*(avg_neg + avg_pos) - 0.5f*fabsf(VX)*(avg_pos-avg_neg);
}

inline __device__ real spatialFluxY(real avg_neg,real avg_pos,real* blockParams) {
   creal VY = blockParams[BlockParams::VYCRD] + (threadIdx.y+0.5f)*blockParams[BlockParams::DVY];
   return 0.5f*VY*(avg_neg + avg_pos) - 0.5f*fabsf(VY)*(avg_pos-avg_neg);
}

inline __device__ real spatialFluxZ(real avg_neg,real avg_pos,real* blockParams,uint z_ind) {
   creal VZ = blockParams[BlockParams::VZCRD] + (z_ind+0.5f)*blockParams[BlockParams::DVZ];
   return 0.5f*VZ*(avg_neg + avg_pos) - 0.5f*fabsf(VZ)*(avg_pos-avg_neg);
}


